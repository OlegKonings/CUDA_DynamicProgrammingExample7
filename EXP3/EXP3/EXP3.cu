#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <utility>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <string>
#include <cmath>
#include <vector>
#include <ctime>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <MMSystem.h>
#pragma comment(lib, "winmm.lib")

using namespace std;

typedef long long ll; 
#define all(c) (c).begin(),(c).end() 
typedef pair<int,int> Pii;
typedef vector<int> Vi;

#define _DTH hipMemcpyDeviceToHost
#define _DTD hipMemcpyDeviceToDevice
#define _HTD hipMemcpyHostToDevice

#define THREADS 64

#define MAXLEN 210//note: change if will be longer strings
#define INF (1<<29)

bool InitMMTimer(UINT wTimerRes);
void DestroyMMTimer(UINT wTimerRes, bool init);

void generate_random_combo_strings(string &s0, string &s1, const int len);

bool eval(const int *cur, const int *goal, const int len,int loc, int up, int down){
	if(loc>len)return false;
	return (((cur[loc]+up-down)%10+10)%10==goal[loc]);
}
inline int _3d_flat(int i, int j, int k, int D1,int D0){return i*D1*D0+j*D0+k;}

int cpu_version(const int *current, const int *goal, const int len){
	const int problem_space=(len+1)*(len+1)*(len+1);
	const int num_bytes=problem_space*sizeof(int);
	int ans=INF;

	int *DP=(int *)malloc(num_bytes);
	bool *A0=(bool *)malloc((len+1)*(len+1)*sizeof(bool));
	bool *A1=(bool *)malloc((len+1)*(len+1)*sizeof(bool));
	
	for(int i=1;i<problem_space;i++){
		DP[i]=INF;
	}

	DP[0]=0;
	for(int i=1;i<=len;i++){

		memset(A0,0,(len+1)*(len+1)*sizeof(bool));
		memset(A1,0,(len+1)*(len+1)*sizeof(bool));

		for(int j=0;j<=len;j++)for(int k=0;k<=len;k++){
			if(eval(current,goal,len,i,j,k))A0[j*(len+1)+k]=true;
		}
		for(int j=0;j<=len;j++)for(int k=0;k<=len;k++){
			if(DP[_3d_flat(i-1,j,k,(len+1),(len+1))]!=INF)
				A1[j*(len+1)+k]=true;
		}

		for(int x=0;x<=len;x++)for(int y=0;y<=len;y++)if(A0[x*(len+1)+y]){
			for(int xx=0;xx<=len;xx++)for(int yy=0;yy<=len;yy++)if(A1[xx*(len+1)+yy]){
				int temp=DP[_3d_flat(i-1,xx,yy,(len+1),(len+1))]+max(0,x-xx)+max(0,y-yy);
				if(temp<DP[_3d_flat(i,x,y,(len+1),(len+1))]){
					DP[_3d_flat(i,x,y,(len+1),(len+1))]=temp;
				}
			}
		}
	}
	for(int i=0;i<=len;i++)for(int j=0;j<=len;j++){
		if(DP[_3d_flat(len,i,j,(len+1),(len+1))]<ans){
			ans=DP[_3d_flat(len,i,j,(len+1),(len+1))];
		}
	}

	free(DP);
	free(A0);
	free(A1);
	return ans;
}

__constant__ int D_cur[MAXLEN];
__constant__ int D_goal[MAXLEN];

__device__ __forceinline__ int D_3d_flat(int i, int j, int k, int D1,int D0){return D0*(i*D1+j)+k;}

__global__ void set_DP(int *D_DP,const int problemspace){
	const int offset=threadIdx.x+blockIdx.x*blockDim.x;
	if(offset<problemspace){
		D_DP[offset]=INF;
	}
}

__global__ void GPU_version(const int ii, int *D_DP, const int len){
	
	const int l=blockIdx.z;
	const int m=blockIdx.y;

	__shared__ int best;

	if(threadIdx.x==0){
		best=D_DP[D_3d_flat(ii-1,l,m,(len+1),(len+1))];
	}

	__syncthreads();

	if(best>=INF)return;

	const int j=threadIdx.x+blockIdx.x*blockDim.x;

	if(j>len)return;

	for(int k=0;k<=len;k++){
		if(((D_cur[ii]+j-k)%10+10)%10==D_goal[ii]){
			atomicMin(&D_DP[D_3d_flat(ii,j,k,(len+1),(len+1))],(best+max(0,j-l)+max(0,k-m)) );

		}
	}
}

__global__ void last_step(const int *D_DP,int *best_val,const int len){
	const int i=threadIdx.x+blockIdx.x*blockDim.x;
	if(i<=len){
		atomicMin(&best_val[0],D_DP[D_3d_flat(len,i,blockIdx.y,(len+1),(len+1))]);
	}
}

int main(){

        srand(time(NULL));
		
		string s0="5390863801527525349142229108298075699798617845613912347987984732789432009090909090904218989432814923";
		string s1="5691764076679014302854836840311218635202200369261121447812739923746784821749837498209099098423788737";

		bool generate_random=true;
		if(generate_random){
			int big_combo_string_size=200;
			s0.clear();
			s1.clear();
			s0.resize(big_combo_string_size,'0');
			s1.resize(big_combo_string_size,'0');
			generate_random_combo_strings(s0,s1,big_combo_string_size);

			cout<<"\nstarting string= "<<s0<<'\n';
			cout<<"target string= "<<s1<<'\n';

		}


		const int s_len=s0.length();
		cout<<"\nLength= "<<s_len<<'\n';
		int *a0=(int *)malloc((s_len+1)*sizeof(int));
		int *a1=(int *)malloc((s_len+1)*sizeof(int));
		a0[0]=a1[0]=-1;
		for(int i=1;i<=s_len;i++){
			a0[i]=int(s0[i-1]-'0');
			a1[i]=int(s1[i-1]-'0');

		}
		int CPU_ans=0,GPU_ans=-1;
		//CPU
		cout<<"\nRunning CPU implementation..\n";
		UINT wTimerRes = 0;
		DWORD CPU_time=0,GPU_time=0;
		bool init = InitMMTimer(wTimerRes);
		DWORD startTime=timeGetTime();
	
		CPU_ans=cpu_version(a0,a1,s_len);

		DWORD endTime = timeGetTime();
		CPU_time=endTime-startTime;
		cout<<"CPU solution timing: "<<CPU_time<< " , answer= "<<CPU_ans<<'\n';
		DestroyMMTimer(wTimerRes, init);
		hipError_t err=hipFree(0);

		err=hipMemcpyToSymbol(HIP_SYMBOL(D_cur),a0,(s_len+1)*sizeof(int));
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipMemcpyToSymbol(HIP_SYMBOL(D_goal),a1,(s_len+1)*sizeof(int));
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		const int problemspace=(s_len+1)*(s_len+1)*(s_len+1);
		const int num_bytes=problemspace*sizeof(int);
		int *D_DP,*best_val;
		err=hipMalloc((void**)&D_DP,num_bytes);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipMalloc((void**)&best_val,sizeof(int));
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		const int num_threads= (s_len>=128) ? 128:64;

		int ii=1,B_val=INF;
		dim3 Grid((s_len+num_threads)/num_threads,(s_len+1),(s_len+1));

		wTimerRes = 0;
		init = InitMMTimer(wTimerRes);
		startTime = timeGetTime();

		set_DP<<<(problemspace+num_threads-1)/num_threads,num_threads>>>(D_DP,problemspace);
		err = hipDeviceSynchronize();
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		err = hipMemset(D_DP,0,sizeof(int));
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		for(;ii<=s_len;ii++){
			GPU_version<<<Grid,num_threads>>>(ii,D_DP,s_len);
			err = hipDeviceSynchronize();
			if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		}

		err=hipMemcpy(best_val,&B_val,sizeof(int),_HTD);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		Grid.z=1;
		last_step<<<Grid,num_threads>>>(D_DP,best_val,s_len);
		err = hipDeviceSynchronize();
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		err=hipMemcpy(&GPU_ans,best_val,sizeof(int),_DTH);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		endTime = timeGetTime();
		GPU_time=endTime-startTime;
		cout<<"CUDA timing: "<<GPU_time<<" , answer= "<<GPU_ans<<'\n';
		DestroyMMTimer(wTimerRes, init);

		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipFree(D_DP);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipFree(best_val);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		free(a0);
		free(a1);

		err=hipDeviceReset();
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
   
        return 0;
}

bool InitMMTimer(UINT wTimerRes){
        TIMECAPS tc;
        if (timeGetDevCaps(&tc, sizeof(TIMECAPS)) != TIMERR_NOERROR) {return false;}
        wTimerRes = min(max(tc.wPeriodMin, 1), tc.wPeriodMax);
        timeBeginPeriod(wTimerRes); 
        return true;
}

void DestroyMMTimer(UINT wTimerRes, bool init){
        if(init)
			timeEndPeriod(wTimerRes);
}

void generate_random_combo_strings(string &s0, string &s1, const int len){
	int r0=0,r1=0;
	for(int i=0;i<len;i++){
		r0=rand()%10;
		r1=rand()%10;
		s0[i]=char('0'+r0);
		s1[i]=char('0'+r1);
	}


}
